#include "hip/hip_runtime.h"
// Original implementation by Mykola Dvornik for mumax2
// Modified for mumax3 by Arne Vansteenkiste, 2013

#include <stdint.h>
#include "float3.h"
#include "constants.h"
#include "amul.h"

extern "C" __global__ void
addoommfslonczewskitorque(float* __restrict__ tx, float* __restrict__ ty, float* __restrict__ tz,
                     float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
                      float* __restrict__ Ms_,      float  Ms_mul,
                      float* __restrict__ jz_,      float  jz_mul,
                      float* __restrict__ px_,      float  px_mul,
                      float* __restrict__ py_,      float  py_mul,
                      float* __restrict__ pz_,      float  pz_mul,
                      float* __restrict__ alpha_,   float  alpha_mul,
                      float* __restrict__ pfix_,     float  pfix_mul,
                      float* __restrict__ pfree_,     float  pfree_mul,
                      float* __restrict__ lambdafix_,  float  lambdafix_mul,
                      float* __restrict__ lambdafree_,  float  lambdafree_mul,
                      float* __restrict__ epsPrime_,float  epsPrime_mul,
                      float* __restrict__ flt_,     float  flt_mul,
                     int N) {

	int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
	if (i < N) {

		float3 m = make_float3(mx[i], my[i], mz[i]);
		float  J = amul(jz_, jz_mul, i);
		float3 p = normalized(vmul(px_, py_, pz_, px_mul, py_mul, pz_mul, i));

        	float  Ms           = amul(Ms_, Ms_mul, i);
        	float  alpha        = amul(alpha_, alpha_mul, i);
        	float  flt          = amul(flt_, flt_mul, i);
        	float  pfix         = amul(pfix_, pfix_mul, i);
        	float  pfree        = amul(pfree_, pfix_mul, i);
		float  lambdafix    = amul(lambdafix_, lambdafix_mul, i);
		float  lambdafree   = amul(lambdafree_, lambdafree_mul, i);
	        float  epsilonPrime = amul(epsPrime_, epsPrime_mul, i);

		if (J == 0.0f || Ms == 0.0f) {
			return;
		}

		float beta    = (HBAR / QE) * (J / (2.0f *flt*Ms) );
		float lambdafix2 = lambdafix * lambdafix;
		float lambdafree2 = lambdafree * lambdafree;
		float lambdafreePlus = sqrt(lambdafree2 + 1.0f);
		float lambdafixPlus = sqrt(lambdafix2 + 1.0f);
		float lambdafreeMinus = sqrt(lambdafree2 - 1.0f);
		float lambdafixMinus = sqrt(lambdafix2 - 1.0f);
		float plus_ratio = lambdafreePlus / lambdafixPlus;
		float minus_ratio = 1.0f;
		if (lambdafreeMinus > 0) {
		   	minus_ratio = lambdafixMinus / lambdafreeMinus;
		}
		// Compute q_plus and q_minus
		float plus_factor = pfix * lambdafix2 * plus_ratio;
		float minus_factor = pfree * lambdafree2 * minus_ratio;
		float q_plus = plus_factor + minus_factor;
		float q_minus = plus_factor - minus_factor;
		float lplus2 = lambdafreePlus * lambdafixPlus;
		float lminus2 = lambdafreeMinus * lambdafixMinus;
		float pdotm = dot(p, m);
		float A_plus = lplus2 + (lminus2 * pdotm);
		float A_minus = lplus2 - (lminus2 * pdotm);
		float epsilon = (q_plus / A_plus) - (q_minus / A_minus);

		float A = beta * epsilon;
		float B = beta * epsilonPrime;

		float gilb     = 1.0f / (1.0f + alpha * alpha);
		float mxpxmFac = gilb * (A - alpha * B);
		float pxmFac   = gilb * (B - alpha * A);

		float3 pxm      = cross(p, m);
		float3 mxpxm    = cross(m, pxm);

		tx[i] += mxpxmFac * mxpxm.x + pxmFac * pxm.x;
		ty[i] += mxpxmFac * mxpxm.y + pxmFac * pxm.y;
		tz[i] += mxpxmFac * mxpxm.z + pxmFac * pxm.z;
	}
}
